
#include "hip/hip_runtime.h"


#include <cstdio>
#include <cstdlib>

static __global__ void memArgsKernel(int *args[2])
{
    auto const localThreadID = threadIdx.x;
    auto const blockSize = blockDim.x;
    auto const blockID = blockIdx.x;
    auto const globalThreadID = blockID * blockSize + localThreadID;

    args[0][globalThreadID] += args[1][globalThreadID];
}

static void TestMemArgs(void)
{
    int* devMem0 = nullptr;
    int* devMem1 = nullptr;
    int* *devArgMem = nullptr;

    constexpr int elemCount = 4096;
    constexpr auto bufferSize = elemCount * sizeof(*devMem0);

    int* mainMem = (int*)malloc(bufferSize);
    if (mainMem == nullptr)
        return;

    for (int i = 0; i < elemCount; i++)
        mainMem[i] = 1;

    do
    {
        auto cudaStatus = hipMalloc(&devMem0, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMalloc(&devMem1, bufferSize);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        int* args[] = { devMem0, devMem1 };

        cudaStatus = hipMalloc(&devArgMem, sizeof(args));
        if (cudaStatus != hipSuccess)
        {
            printf("hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devMem0, mainMem, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devMem1, mainMem, bufferSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        cudaStatus = hipMemcpy(devArgMem, args, sizeof(args), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        hipDeviceProp_t props{ };
        cudaStatus = hipGetDeviceProperties(&props, 0);
        if (cudaStatus != hipSuccess)
        {
            printf("hipGetDeviceProperties failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        auto const threadCount = props.maxThreadsPerBlock;
        auto const blockCount = elemCount / threadCount;
        memArgsKernel <<< blockCount, threadCount >>> (devArgMem);

        cudaStatus = hipMemcpy(mainMem, devMem0, bufferSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            printf("hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        // Verify the result
        for (int i = 0; i < elemCount; i++)
        {
            if (mainMem[i] != 2)
            {
                printf("Error occurred @ %d\n", i);
                break;
            }
        }

    } while (false);

    free(mainMem);

    if (devMem0 != nullptr)
        hipFree(devMem0);
    if (devMem1 != nullptr)
        hipFree(devMem1);
    if (devArgMem != nullptr)
        hipFree(devArgMem);
}

int main(void)
{
    auto cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        printf("hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }

    TestMemArgs();

    puts("Test completed!!");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        printf("hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
        return 0;
    }
}

